#include "networklayer.h"
#include <limits>
#include <thrust/device_vector.h>

NetworkLayer::NetworkLayer(std::string n, ActivationType a, const std::vector<unsigned> &s, NetworkLayer *pl)
	:_layerName(n), _activationType(a), _layerSize(s), _prevLayer(pl)
{
	std::cout << "NetworkLayer::NetworkLayer" << std::endl;
	
	int lts = 0;
	for(unsigned i=0; i<s.size(); ++i)
		lts += s[i];
	_activations = std::vector<float>(lts, 333);
}

std::string NetworkLayer::layerName() const
{
	//std::cout << "NetworkLayer::layerName" << std::endl;
	return _layerName;
}

ActivationType NetworkLayer::activationType() const
{
	//std::cout << "NetworkLayer::activationType" << std::endl;
	
	return _activationType;
}

std::vector<unsigned> NetworkLayer::layerSize() const
{
	//std::cout << "NetworkLayer::layerSize" << std::endl;
	return _layerSize;
}

BiasVertex *NetworkLayer::biasVertex() const
{
	//std::cout << "NetworkLayer::biasVertex" << std::endl;

	return _bias;
}

std::vector<float> NetworkLayer::activations() const
{
	//std::cout << "NetworkLayer::activations" << std::endl;

	return _activations;
}

//----------------------------------------------------------------------------------------------------------------------------------------//

InputLayer::InputLayer(std::string ln, ActivationType at, const std::vector<unsigned> &ls, NetworkLayer *pl)
	:NetworkLayer(ln, at, ls, pl)//, _activations(std::vector<float>(ls[0]*ls[1]*ls[2] + 1))
{
	std::cout << "InputLayer::InputLayer" << std::endl;

	int layerTotalSize = _layerSize[0]*_layerSize[1]*_layerSize[2];
	_activations = std::vector<float>(layerTotalSize + 1);
	_vertices = new Tensor3d<Input3dVertex*>(_layerSize[0], _layerSize[1], _layerSize[2]);
	_bias = new BiasVertex(&_activations[layerTotalSize], 0);
	for(unsigned i=0; i<_layerSize[0]; ++i)
		for(unsigned j=0; j<_layerSize[1]; ++j)
			for(unsigned k=0; k<_layerSize[2]; ++k)
			{
				int actIndex = i*_layerSize[1]*_layerSize[2] + j*_layerSize[2] + k;
				(*_vertices)[i][j][k] = new Input3dVertex(&_activations[actIndex], 0);
			}
}

NetworkLayer::LayerType InputLayer::layerType() const
{
	//std::cout << "InputLayer::layerType" << std::endl;
	
	return NetworkLayer::INPUT;
}

void InputLayer::forwardProp(PropagationType p)
{
	//std::cout << "InputLayer::forwardProp" << std::endl;

	unsigned height = _layerSize[0];
	unsigned width = _layerSize[1];
	unsigned depth = _layerSize[2];

	for(unsigned i=0; i<height; ++i)
		for(unsigned j=0; j<width; ++j)
			for(unsigned k=0; k<depth; ++k)
				(*_vertices)[i][j][k]->setActivation((*_state)[i*width*depth + j*depth + k]);

	//std::cout << "InputLayer::forwardProp" << std::endl;
}

void InputLayer::backProp(unsigned expNum, const std::vector<double> &action, double delta)
{
	//std::cout << "InputLayer::backProp" << std::endl;	
}

/*
void InputLayer::forwardPropTarget()
{
	std::cout << "InputLayer::forwardPropTarget" << std::endl;

	unsigned length = _layerSize[0];
	unsigned width = _layerSize[1];
	unsigned depth = _layerSize[2];
	for(unsigned i=0; i<length; ++i)
		for(unsigned j=0; j<width; ++j)
			for(unsigned k=0; k<depth; ++k)
				(*_vertices)[i][j][k]->setActivation((*_state)[i*width*depth + j*depth + k]);
	
}
*/

Tensor3d<Input3dVertex*>* InputLayer::vertices() const
{
	//std::cout << "InputLayer::vertices" << std::endl;

	return _vertices;
}


void InputLayer::setState(vizdoom::BufferPtr s)
{
	//std::cout << "InputLayer::setState" << std::endl;

	_state = s;
}



Conv3dLayer::Conv3dLayer(std::string ln, ActivationType at, std::vector<unsigned> ls, NetworkLayer *prevLayer, unsigned fdi, unsigned fde, unsigned fs)
	:NetworkLayer(ln, at, ls, prevLayer), _filterDim(fdi), _filterDepth(fde), _filterStride(fs)
{
	std::cout << "Conv3dLayer::Conv3dLayer" << std::endl;

	// Cannot cast Tensor3d<Conv3dVertex*>* / Tensor3d<Pool3dVertex*>* to Tensor3d<Vertex*>* since overload of pointer assignment operator is not allowed. The result
	// of this is that *prevVertices is not pointing to original Tensor3d<Conv3dVertex*>/Tensor3d<Pool3dVertex*> struct but to a new struct whcih holds pointers
	// identical to the ones in the original struct. This new struct has to be deleted upon exit of this constructor to avoid memory leak, but carefuly not to
	// delete the vertices to which pointers inside this struct point to.
    Tensor3d<Vertex*> *prevVertices = nullptr;
	if(prevLayer->layerType() == NetworkLayer::INPUT)
		prevVertices = new Tensor3d<Vertex*>(((InputLayer*)prevLayer)->vertices());
	else if(prevLayer->layerType() == NetworkLayer::CONV)
		//prevVertices = ((Conv3dLayer*)prevLayer)->vertices();
		prevVertices = new Tensor3d<Vertex*>(((Conv3dLayer*)prevLayer)->vertices());
	else if(prevLayer->layerType() == NetworkLayer::MAX_POOL)
		//prevVertices = ((Pool3dLayer*)prevLayer)->vertices();
		prevVertices = new Tensor3d<Vertex*>(((Pool3dLayer*)prevLayer)->vertices());
	else
	{
		std::cout << "Unexpected previous layer type. Previous layer should be INPUT, CONV or POOL." << std::endl;
		std::exit(1);
	}

	int layerTotalSize = _layerSize[0]*_layerSize[1]*_layerSize[2];
	int filterTotalSize = _filterDim*_filterDim*_filterDepth;
	// Total number of weights is 3d-filter size times number of filters plus one bias for each filter. That is for each filter (depth dimension) there are
	// (filterTotalSize + 1) weights. (filterTotalSize + 1)*_layerSize[2]
	_weights = std::vector<float>(filterTotalSize*_layerSize[2] + _layerSize[2]);
	_cachedWeights = std::vector<float>(filterTotalSize*_layerSize[2] + _layerSize[2]);
	_TDUpdates = std::vector<float>(filterTotalSize*_layerSize[2] + _layerSize[2]);
	_outGrads = std::vector<float>((filterTotalSize + 1)*layerTotalSize);
	_dotProducts = std::vector<float>(layerTotalSize);
	_activations = std::vector<float>(layerTotalSize + 1);

	_vertices = new Tensor3d<Conv3dVertex*>(_layerSize[0], _layerSize[1], _layerSize[2]);
	_bias = new BiasVertex(&_activations[layerTotalSize], 0);
	for(unsigned i=0; i<_layerSize[0]; ++i)
		for(unsigned j=0; j<_layerSize[1]; ++j)
			for(unsigned k=0; k<_layerSize[2]; ++k)
			{
				int vIndex = i*_layerSize[1]*_layerSize[2] + j*_layerSize[2] + k;
				//Tensor3d<WeightedEdge*> *inputEdges = new Tensor3d<WeightedEdge*>(_filterDim, _filterDim, _filterDepth);
				Tensor1d<WeightedEdge*> *inputEdges = new Tensor1d<WeightedEdge*>(filterTotalSize + 1);
				Conv3dVertex *v = new Relu3dUnit(&_activations[vIndex], 0.0f, &_dotProducts[vIndex], 0.0f, inputEdges);
				for(unsigned h=0; h<_filterDim; ++h)
					for(unsigned w=0; w<_filterDim; ++w)
						for(unsigned d=0; d<_filterDepth; ++d)
						{
							int eIndex = k*(filterTotalSize + 1) + h*_filterDim*_filterDepth + w*_filterDepth + d;
							int gIndex = vIndex*(filterTotalSize + 1) + h*_filterDim*_filterDepth + w*_filterDepth + d;
							WeightedEdge *e = new WeightedEdge((*prevVertices)[i+h][j+w][d], /*v,*/ &_outGrads[gIndex], &_weights[eIndex], &_TDUpdates[eIndex]);
							//(*inputEdges)[h][w][d] = e;
							(*inputEdges)[h*_filterDim*_filterDepth + w*_filterDepth + d] = e;
							(*prevVertices)[i+h][j+w][d]->addOutputEdge(e);
						}

				int eIndex = k*(filterTotalSize + 1) + filterTotalSize;
				int gIndex = vIndex*(filterTotalSize + 1) + filterTotalSize;
				WeightedEdge *e = new WeightedEdge(biasVertex(), /*v,*/ &_outGrads[gIndex], &_weights[eIndex], &_TDUpdates[eIndex]);
				(*inputEdges)[filterTotalSize] = e;
				biasVertex()->addOutputEdge(e);
				(*_vertices)[i][j][k] = v;
			}
}

NetworkLayer::LayerType Conv3dLayer::layerType() const
{
	//std::cout << "Conv3dLayer::layerType" << std::endl;
	
	return NetworkLayer::CONV;
}

struct Conv3dTransform{
	float *_input;
	float *_weights;
	int _filterDim;
	int _filterDepth;
	int _inputHeight;
	int _inputWidth;
	int _inputDepth;
	int _layerHeight;
	int _layerWidth;
	int _layerDepth;

	Conv3dTransform(float *i, float *w, int fdi, int fde, int ih, int iw, int id, int lh, int lw, int ld)
		:_input(i), _weights(w), _filterDim(fdi), _filterDepth(fde), 
		_inputHeight(ih), _inputWidth(iw), _inputDepth(id), 
		_layerHeight(lh), _layerWidth(lw), _layerDepth(ld)
	{
			
	}
	__host__ __device__ thrust::tuple<float, float> operator()(size_t vidx)
	{
		// vidx = i*outputWidth*outputDepth + j*outputDepth + k
		int i = vidx/(_layerWidth*_layerDepth);
		int jk = vidx - (i*_layerWidth*_layerDepth);
		int j = jk/_layerDepth;
		int k = jk - (j*_layerDepth);

		int widx = k*(_filterDim*_filterDim*_filterDepth + 1);
		// Input vertex index should be the same as the output one? No, this was a bug. The input vertex index is always in the
		// first input channel. Thats why there is no + k at the end of right hand side of the expression bellow.
		int iidx = i*_inputWidth*_inputDepth + j*_inputDepth;
		float dotProduct = 0.0f;
		float activation;
		for(unsigned h=0; h<_filterDim; ++h)
			for(unsigned w=0; w<_filterDim; ++w)
				for(unsigned d=0; d<_filterDepth; ++d)
				{
					//int wx = k*(filterTotalSize + 1) + h*_filterDim*_filterDepth + w*_filterDepth + d;
					int wx = widx + h*_filterDim*_filterDepth + w*_filterDepth + d;
					int ix = iidx + h*_inputWidth*_inputDepth + w*_inputDepth + d;
					dotProduct += _input[ix]*_weights[wx];
				}

		dotProduct += _weights[widx + _filterDim*_filterDim*_filterDepth];
		activation = (dotProduct > 0)? dotProduct : 0;
		return thrust::make_tuple(dotProduct, activation);
	}
};

void Conv3dLayer::forwardProp(PropagationType p)
{
	//std::cout << "Conv3dLayer::forwardProp" << std::endl;

	int layerSize = _layerSize[0]*_layerSize[1]*_layerSize[2];
	int inputHeight = _prevLayer->layerSize()[0];
	int inputWidth = _prevLayer->layerSize()[1];
	int inputDepth = _prevLayer->layerSize()[2];

	std::vector<float> act = _prevLayer->activations();
	thrust::device_vector<float> input(act.begin(), act.end());
	thrust::device_vector<float> weights(_weights.size());
	if(p == PREDICTION)
		thrust::copy(_weights.begin(), _weights.end(), weights.begin());
	else if(p == TARGET)
		thrust::copy(_cachedWeights.begin(), _cachedWeights.end(), weights.begin());
	
	thrust::device_vector<float> dotProducts(layerSize);
	thrust::device_vector<float> activations(layerSize);

	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(layerSize), 
		thrust::make_zip_iterator(thrust::make_tuple(dotProducts.begin(), activations.begin())), 
		Conv3dTransform(thrust::raw_pointer_cast(input.data()), thrust::raw_pointer_cast(weights.data()), 
			_filterDim, _filterDepth, inputHeight, inputWidth, inputDepth, _layerSize[0], _layerSize[1], _layerSize[2]));
	hipDeviceSynchronize();

	// It looks like it works great with std::vector<float> as output vector of thrust::copy. 
	// Maybe try thrust::host_vector<float> as member vectors as well. Also, make sure location of output vector is not changed.
	thrust::copy(dotProducts.begin(), dotProducts.end(), _dotProducts.begin());
	thrust::copy(activations.begin(), activations.end(), _activations.begin());
	
	//std::cout << "Conv3dLayer::forwardProp" << std::endl;
}

struct Conv3dBack{
	float *_inGrad;
	float *_dotProducts;
	float *_prevAct;
	float *_weights;
	float *_outGrads;
	float _delta;
	unsigned _filterDim;
	unsigned _filterDepth;
	unsigned _layerHeight;
	unsigned _layerWidth;
	unsigned _layerDepth;
	unsigned _prevLayerWidth;
	unsigned _prevLayerDepth;

	Conv3dBack(float *ig, float *dp, float *pa, float *w, float *og, float d, 
			unsigned fdi, unsigned fde, unsigned lh, unsigned lw, unsigned ld, unsigned plw, unsigned pld)
		:_inGrad(ig), _dotProducts(dp), _prevAct(pa), _weights(w), _outGrads(og), _delta(d),
		_filterDim(fdi), _filterDepth(fde), _layerHeight(lh), _layerWidth(lw), _layerDepth(ld), _prevLayerWidth(plw), _prevLayerDepth(pld)
	{}
	__host__ __device__ float operator()(size_t widx)
	{
		// actGrad = gradRelu(inGrad, dotProduct)
		// outGrad = weight*actGrad = weight*gradRelu(inGrad, dotProduct)
		// weightGrad = prevAct*actGrad = prevAct*gradRelu(inGrad, dotProduct) 
		// tdUpdate = weightGrad*delta

		int weightsTotalSize = _filterDim*_filterDim*_filterDepth + 1;
		// The channel in which weight widx is located.
		int k = widx/weightsTotalSize;
		// Location of widx and its edge in input edges of each vertex.
		int r = widx - k*weightsTotalSize;

		int h = r/_filterDim*_filterDepth;
		int wd = r - h*_filterDim*_filterDepth;
		int w = wd/_filterDepth;
		int d = wd - w*_filterDepth;

		// Fixating the k-th channel because weight widx is common only for that channel.
		float tdUpdate = 0;
		for(unsigned i=0; i<_layerHeight; ++i)
			for(unsigned j=0; j<_layerWidth; ++j)
			{
				int vidx = i*_layerWidth*_layerDepth + j*_layerDepth + k;
				float actGrad = (_dotProducts[vidx] >= 0)? _inGrad[vidx] : 0;

				int gidx = vidx*weightsTotalSize + r;
				_outGrads[gidx] = _weights[widx]*actGrad;

				// If current weight widx is bias then previous activation is one.
				if(r == weightsTotalSize - 1)
					tdUpdate += actGrad*_delta;
				else
				{
					// This is the vertex from previous layer that is connected via edge of widx. First is calculated the top left vertex 
					// in previous layer (vidx + skipped vertices because of filter size) and then adding the precise location of wanted
					// vertex in convolution block with respect to top left corner. It is easier to understand on a drawing.
					int aidx = vidx + i*(_filterDim - 1)*_prevLayerDepth + h*_prevLayerWidth*_prevLayerDepth + w*_prevLayerDepth + d;
					tdUpdate += _prevAct[aidx]*actGrad*_delta;					
				}
			}

		return tdUpdate;
	}
};

void Conv3dLayer::backProp(unsigned expNum, const std::vector<double> &action, double delta)
{
	//std::cout << "Conv3dLayer::backProp" << std::endl;

	int layerTotalSize = _layerSize[0]*_layerSize[1]*_layerSize[2];
	std::vector<float> grad(layerTotalSize);
	for(unsigned i=0; i<_layerSize[0]; ++i)
		for(unsigned j=0; j<_layerSize[1]; ++j)
			for(unsigned k=0; k<_layerSize[2]; ++k)
			{
				int gidx = i*_layerSize[1]*_layerSize[2] + j*_layerSize[2] + k;
				std::vector<Edge*> outputEdges = (*_vertices)[i][j][k]->outputEdges();
				float inGrad = 0;
				for(int l=0; l<outputEdges.size(); ++l)
					inGrad += outputEdges[l]->outGrad();
		
				grad[gidx] = inGrad;
			}

	thrust::device_vector<float> inGrads(grad.begin(), grad.end());
	thrust::device_vector<float> dotProducts(_dotProducts.begin(), _dotProducts.end());
	//thrust::device_vector<float> actGrad(_activations.size());

	std::vector<float> act = _prevLayer->activations();
	thrust::device_vector<float> prevAct(act.begin(), act.end());
	thrust::device_vector<float> weights(_weights.begin(), _weights.end());

	thrust::device_vector<float> outGrads(_outGrads.size());
	thrust::device_vector<float> tdUpdates(_TDUpdates.size());

	std::vector<unsigned> prevLayerSize = _prevLayer->layerSize();
	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(_weights.size()), tdUpdates.begin(), 
		Conv3dBack(thrust::raw_pointer_cast(inGrads.data()), thrust::raw_pointer_cast(dotProducts.data()), 
			thrust::raw_pointer_cast(prevAct.data()), thrust::raw_pointer_cast(weights.data()), thrust::raw_pointer_cast(outGrads.data()), delta,
			_filterDim, _filterDepth, _layerSize[0], _layerSize[1], _layerSize[2], prevLayerSize[1], prevLayerSize[2]));
	hipDeviceSynchronize();

	thrust::copy(outGrads.begin(), outGrads.end(), _outGrads.begin());
	thrust::copy(tdUpdates.begin(), tdUpdates.end(), _TDUpdates.begin());		
}

void Conv3dLayer::cacheWeights()
{
	//std::cout << "Conv3dLayer::cacheWeights" << std::endl;

	_cachedWeights = _weights;
}

/*thrust::device_ptr<float> Conv3dLayer::weightsToDevice() const
{
	//#include <thrust/device_ptr.h>
	//#include <thrust/fill.h>
	//#include <hip/hip_runtime.h>

	size_t N = 10;
	int layerSize = _layerSize[0]*_layerSize[1]*_layerSize[2];

	// allocate pointer to device memory
	float *rawPtr;
	hipMalloc((void **) &rawPtr, N*sizeof(float));

	// wrap raw pointer with a device_ptr
	thrust::device_ptr<float> devPtr(rawPtr);

	// copy memory to a new device_vector (which automatically allocates memory)
	//thrust::device_vector<float> vec(devPtr, devPtr + N);

	// use vec
	
	// free user-allocated memory
	//hipFree(rawPtr);
	
	return devPtr;
}*/

unsigned Conv3dLayer::filterDim() const
{
	//std::cout << "Conv3dLayer::filterDim" << std::endl;	

	return _filterDim;
}

unsigned Conv3dLayer::filterDepth() const
{	
	//std::cout << "Conv3dLayer::filterDepth" << std::endl;	

	return _filterDepth;
}

unsigned Conv3dLayer::filterStride() const
{
	//std::cout << "Conv3dLayer::filterStride" << std::endl;
	
	return _filterStride;
}

std::vector<float> Conv3dLayer::weights() const
{
	std::cout << "Conv3dLayer::weights" << std::endl;

	return _weights;
}

std::vector<float> Conv3dLayer::dotProducts() const
{
	std::cout << "Conv3dLayer::dotProducts" << std::endl;

	return _dotProducts;
}

Tensor3d<Conv3dVertex*> *Conv3dLayer::vertices() const
{
	//std::cout << "Conv3dLayer::vertices" << std::endl;
	return _vertices;
}

void Conv3dLayer::setWeights(const std::vector<float> &w)
{
	std::cout << "Conv3dLayer::setWeights" << std::endl;

	_weights.clear();
	std::copy(w.begin(), w.end(), std::back_inserter(_weights));
}

//------------------------------------------------------------------------------------------------------------------------------------------------------------//
Pool3dLayer::Pool3dLayer(std::string ln, ActivationType at, std::vector<unsigned> ls, NetworkLayer *prevLayer, unsigned pdi, unsigned ps)
    :NetworkLayer(ln, at, ls, prevLayer), _poolDim(pdi), _poolStride(ps)
{
	std::cout << "Pool3dLayer::Pool3dLayer" << std::endl;

	if(prevLayer->layerType() != NetworkLayer::CONV)
	{
		std::cout << "Unexpected previous layer type. Previous layer must be CONV." << std::endl;
		std::exit(1);
	}

	Tensor3d<Conv3dVertex*> *prevVertices = ((Conv3dLayer*)prevLayer)->vertices();
	int layerTotalSize = _layerSize[0]*_layerSize[1]*_layerSize[2];
	_outGrads = std::vector<float>(_poolDim*_poolDim*layerTotalSize);
	_activations = std::vector<float>(layerTotalSize + 1);
	_vertices = new Tensor3d<Pool3dVertex*>(_layerSize[0], _layerSize[1], _layerSize[2]);
	_bias = new BiasVertex(&_activations[layerTotalSize], 0);
	for(unsigned i=0; i<_layerSize[0]; ++i)
		for(unsigned j=0; j<_layerSize[1]; ++j)
			for(unsigned k=0; k<_layerSize[2]; ++k)
			{
				int vIndex = i*_layerSize[1]*_layerSize[2] + j*_layerSize[2] + k;
				Tensor2d<UnweightedEdge*> *inputEdges = new Tensor2d<UnweightedEdge*>(_poolDim, _poolDim);
				MaxPool3dUnit *v = new MaxPool3dUnit(&_activations[vIndex], 0, inputEdges);
				for(unsigned h=0; h<_poolDim; ++h)
					for(unsigned w=0; w<_poolDim; ++w)
					{
						int gIndex = vIndex*_poolDim*_poolDim + h*_poolDim + w;
						UnweightedEdge *e = new UnweightedEdge((*prevVertices)[i+h][j+w][k]/*, v*/, &_outGrads[gIndex]);
						(*inputEdges)[h][w] = e;
						(*prevVertices)[i+h][j+w][k]->addOutputEdge(e);
					}				

				(*_vertices)[i][j][k] = v;
			}
}

NetworkLayer::LayerType Pool3dLayer::layerType() const
{
	//std::cout << "Pool3dLayer::layerType" << std::endl;
	
	return NetworkLayer::MAX_POOL;
}

struct Pool3dTransform{
	float *_input;
	int _poolDim;
	int _inputHeight;
	int _inputWidth;
	int _inputDepth;
	int _layerHeight;
	int _layerWidth;
	int _layerDepth;

	Pool3dTransform(float *i, int pdi, int ih, int iw, int id, int lh, int lw, int ld)
		:_input(i), _poolDim(pdi), 
		_inputHeight(ih), _inputWidth(iw), _inputDepth(id), 
		_layerHeight(lh), _layerWidth(lw), _layerDepth(ld){}
	__host__ __device__  float operator()(size_t vidx)
	{
		// vidx = i*outputWidth*outputDepth + j*outputDepth + k
		int i = vidx/(_layerWidth*_layerDepth);
		int jk = vidx - (i*_layerWidth*_layerDepth);
		int j = jk/_layerDepth;
		int k = jk - (j*_layerDepth);

		// Input vertex index should be the same as the output one? Yes, because unlike convolution, 
		// pooling is applied to each channel independently.
		int iidx = i*_inputWidth*_inputDepth + j*_inputDepth + k;
		float activation = _input[iidx];
		for(unsigned h=0; h<_poolDim; ++h)
			for(unsigned w=0; w<_poolDim; ++w)
			{	
				int ix = iidx + h*_inputWidth*_inputDepth + w*_inputDepth;
				if(_input[ix] > activation)
					activation = _input[ix];
			}	
			
		return activation;
	}

};

void Pool3dLayer::forwardProp(PropagationType p)
{
	//std::cout << "Pool3dLayer::forwardProp" << std::endl;

	int layerSize = _layerSize[0]*_layerSize[1]*_layerSize[2];
	int inputHeight = _prevLayer->layerSize()[0];
	int inputWidth = _prevLayer->layerSize()[1];
	int inputDepth = _prevLayer->layerSize()[2];

	std::vector<float> act = _prevLayer->activations();
	thrust::device_vector<float> input(act.begin(), act.end());
	thrust::device_vector<float> activations(layerSize);

	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(layerSize), 
		activations.begin(), Pool3dTransform(thrust::raw_pointer_cast(input.data()), _poolDim, //_poolDepth, 
			inputHeight, inputWidth, inputDepth, _layerSize[0], _layerSize[1], _layerSize[2]));
	hipDeviceSynchronize();

	// It looks like it works great with std::vector<float> as output vector of thrust::copy. 
	// Maybe try thrust::host_vector<float> as member vector as well. Also, make sure location of output vector is not changed.
	thrust::copy(activations.begin(), activations.end(), _activations.begin());
	//std::cout << "Pool3dLayer::forwardProp" << std::endl;
}

struct Pool3dBack{
	float *_inGrad;
	float *_prevAct;
	unsigned _poolDim;
	unsigned _layerHeight;
	unsigned _layerWidth;
	unsigned _layerDepth;

	Pool3dBack(float *ig, float *pa, unsigned pd, unsigned lh, unsigned lw, unsigned ld)
		:_inGrad(ig), _prevAct(pa), _poolDim(pd), _layerHeight(lh), _layerWidth(lw), _layerDepth(ld)
	{}
	__host__ __device__ float operator()(size_t eidx)
	{
		// outGrad = gradPool(inGrad, prevAct)

		int vidx = eidx/(_poolDim*_poolDim);
		int i = vidx/(_layerWidth*_layerDepth);
		//int jk = vidx - i*_layerWidth*_layerDepth;
		//int j = jk/_layerDepth;
		//int k = jk - j*_layerDepth;

		int hwp = eidx - vidx*_poolDim*_poolDim;
		int hp = hwp/_poolDim;
		int wp = hwp - hp*_poolDim;
		
		int aidx = vidx + i*(_poolDim - 1)*_layerDepth;
		unsigned eix = aidx + hp*_poolDim*_layerDepth + wp*_layerDepth;
		float outGrad = _inGrad[vidx];
		for(unsigned h=0; h<_poolDim; ++h)
			for(unsigned w=0; w<_poolDim; ++w)
			{
				unsigned aix = aidx + h*_poolDim*_layerDepth + w*_layerDepth;
				if(_prevAct[aix] > _prevAct[eix])
					return 0;
			}

		return outGrad;
	}
};


void Pool3dLayer::backProp(unsigned expNum, const std::vector<double> &action, double delta)
{
	//std::cout << "Pool3dLayer::backProp" << std::endl;

	int layerTotalSize = _layerSize[0]*_layerSize[1]*_layerSize[2];
	std::vector<float> grad(layerTotalSize);
	for(unsigned i=0; i<_layerSize[0]; ++i)
		for(unsigned j=0; j<_layerSize[1]; ++j)
			for(unsigned k=0; k<_layerSize[2]; ++k)
			{
				int gidx = i*_layerSize[1]*_layerSize[2] + j*_layerSize[2] + k;
				std::vector<Edge*> outputEdges = (*_vertices)[i][j][k]->outputEdges();
				float inGrad = 0;
				for(int l=0; l<outputEdges.size(); ++l)
					inGrad += outputEdges[l]->outGrad();
		
				grad[gidx] = inGrad;
			}

	thrust::device_vector<float> inGrads(grad.begin(), grad.end());
	//thrust::device_vector<float> dotProducts(_dotProducts.begin(), _dotProducts.end());
	//thrust::device_vector<float> actGrad(_activations.size());

	std::vector<float> act = _prevLayer->activations();
	thrust::device_vector<float> prevAct(act.begin(), act.end());
	//thrust::device_vector<float> weights(_weights.begin(), _weights.end());

	thrust::device_vector<float> outGrads(_outGrads.size());
	//thrust::device_vector<float> tdUpdates(_TDUpdates.size());

	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(outGrads.size()), outGrads.begin(), 
		Pool3dBack(thrust::raw_pointer_cast(inGrads.data()), thrust::raw_pointer_cast(prevAct.data()), _poolDim, _layerSize[0], _layerSize[1], _layerSize[2]));
	hipDeviceSynchronize();

	thrust::copy(outGrads.begin(), outGrads.end(), _outGrads.begin());		
}

unsigned Pool3dLayer::poolDim() const
{
	//std::cout << "Pool3dLayer::poolDim" << std::endl;	

	return _poolDim;
}

Tensor3d<Pool3dVertex*>* Pool3dLayer::vertices() const
{
	//std::cout << "Pool3dLayer::vertices" << std::endl;
	
	return _vertices;
}

//------------------------------------------------------------------------------------------------------------------------------------------------------------//
DenseLayer::DenseLayer(std::string ln, ActivationType at, std::vector<unsigned> ls, NetworkLayer *prevLayer, unsigned hu)
	:NetworkLayer(ln, at, ls, prevLayer), _numHiddenUnits(hu)
{
	std::cout << "DenseLayer::DenseLayer" << std::endl;

	std::vector<unsigned> prevLayerSize = prevLayer->layerSize();
	int prevTotalSize = 1;
	for(unsigned i=0; i<prevLayerSize.size(); ++i)
		prevTotalSize *= prevLayerSize[i];

	std::vector<unsigned> curLayerSize = layerSize();
	_weights = std::vector<float>((prevTotalSize + 1)*_numHiddenUnits);
	_cachedWeights = std::vector<float>((prevTotalSize + 1)*_numHiddenUnits);
	_TDUpdates = std::vector<float>((prevTotalSize + 1)*_numHiddenUnits);
	_outGrads = std::vector<float>((prevTotalSize + 1)*_numHiddenUnits);
	_dotProducts = std::vector<float>(_numHiddenUnits);
	_activations = std::vector<float>(_numHiddenUnits + 1);

	_vertices = new Tensor1d<Dense1dVertex*>(curLayerSize[0]);
	_bias = new BiasVertex(&_activations[_numHiddenUnits], 0);
	
	BiasVertex *prevBias = prevLayer->biasVertex();
	for(unsigned i=0; i<curLayerSize[0]; ++i)
	{
		Dense1dVertex *v;
		if(prevLayer->layerType() == NetworkLayer::FC)
		{
			Tensor1d<Dense1dVertex*> *prevVertices = ((DenseLayer*)prevLayer)->vertices();
            std::vector<unsigned> prevLayerSize = prevLayer->layerSize();
			Tensor1d<WeightedEdge*> *inputEdges = new Tensor1d<WeightedEdge*>(prevLayerSize[0] + 1);
			v = new Relu1dUnit(&_activations[i], 0, &_TDUpdates[i], inputEdges);
            for(unsigned j=0; j<prevLayerSize[0]; ++j)
			{
				int eIndex = i*(prevTotalSize + 1) + j;
				Vertex *u = (*prevVertices)[j];
				WeightedEdge *e = new WeightedEdge(u, /*v,*/ &_outGrads[eIndex], &_weights[eIndex], &_TDUpdates[eIndex]);
				(*inputEdges)[j] = e;
				u->addOutputEdge(e);
			}
		
			int eIndex = i*(prevTotalSize + 1) + prevTotalSize;
			WeightedEdge *e = new WeightedEdge(prevBias, /*v,*/ &_outGrads[eIndex], &_weights[eIndex], &_TDUpdates[eIndex]);
			(*inputEdges)[prevTotalSize] = e;
			prevBias->addOutputEdge(e);

		}
	    else
		{
			// Same issue as in Conv3dLayer::Conv3dLayer. Will have to decide at some point whether should prevVertices be used as structs or pointers to structs. In
			// case pointers are NOT used then consider replacing pointers to vertices structs with only structs in Conv3dLayer and DenseLayer.
	        Tensor3d<Vertex*> *prevVertices = nullptr;
	        if(prevLayer->layerType() == NetworkLayer::CONV)
	            //prevVertices = ((Conv3dLayer*)prevLayer)->vertices();
				prevVertices = new Tensor3d<Vertex*>(((Conv3dLayer*)prevLayer)->vertices());
	        else if(prevLayer->layerType() == NetworkLayer::MAX_POOL)
	        	//prevVertices = ((Pool3dLayer*)prevLayer)->vertices();
				prevVertices = new Tensor3d<Vertex*>(((Pool3dLayer*)prevLayer)->vertices());
        	else
		    {   
				std::cout << "Unexpected previous layer type. Previous layer should be CONV, POOL or FC." << std::endl;
				std::exit(1);
			}

			std::vector<unsigned> prevLayerSize = prevLayer->layerSize();
			Tensor1d<WeightedEdge*> *inputEdges = new Tensor1d<WeightedEdge*>(prevTotalSize + 1);
			v = new Relu1dUnit(&_activations[i], 0, &_dotProducts[i], inputEdges);
			for(unsigned h=0; h<prevLayerSize[0]; ++h)
				for(unsigned w=0; w<prevLayerSize[1]; ++w)
            		for(unsigned d=0; d<prevLayerSize[2]; ++d)
                	{
						int eIndex = i*(prevTotalSize + 1) + h*prevLayerSize[1]*prevLayerSize[2] + w*prevLayerSize[2] + d;
	           			WeightedEdge *e = new WeightedEdge((*prevVertices)[h][w][d], /*v,*/ &_outGrads[eIndex], &_weights[eIndex], &_TDUpdates[eIndex]);
						(*inputEdges)[h*prevLayerSize[1]*prevLayerSize[2] + w*prevLayerSize[2] + d] = e;
						(*prevVertices)[h][w][d]->addOutputEdge(e);
					}

			int eIndex = i*(prevTotalSize + 1) + prevTotalSize;
	        WeightedEdge *e = new WeightedEdge(prevBias, /*v,*/ &_outGrads[eIndex], &_weights[eIndex], &_TDUpdates[eIndex]);
			(*inputEdges)[prevTotalSize] = e;
			prevBias->addOutputEdge(e);
		}

		(*_vertices)[i] = v;
	}
}

NetworkLayer::LayerType DenseLayer::layerType() const
{
	//std::cout << "DenseLayer::layerType" << std::endl;
	
	return NetworkLayer::FC;
}

struct Dense1dTransform{
	float *_input;
	float *_weights;
	int _inputHeight;
	int _layerHeight;
	int _layerWidth;
	int _layerDepth;

	Dense1dTransform(float *i, float *w, int ih, int lh, int lw, int ld)
		:_input(i), _weights(w),
		_inputHeight(ih), 
		_layerHeight(lh), _layerWidth(lw), _layerDepth(ld){}
	__host__ __device__ thrust::tuple<float, float> operator()(size_t vidx)
	{
		// inputSize = inputHeight*inputWidth*inputDepth + bias
		int inputTotalSize = _inputHeight + 1;
		int widx = vidx*inputTotalSize;
		float dotProduct = 0.0f;
		float activation;
		for(unsigned h=0; h<inputTotalSize; ++h)
		{
			int wx = widx + h;
			dotProduct += _input[h]*_weights[wx];
		}
		
		activation = (dotProduct > 0)? dotProduct : 0;
		return thrust::make_tuple(dotProduct, activation);
	}
};

void DenseLayer::forwardProp(PropagationType p)
{
	//std::cout << "DenseLayer::forwardProp" << std::endl;

	int layerSize = _layerSize[0];
	int prevLayerSize;
	if(_prevLayer->layerType() == NetworkLayer::FC)
		prevLayerSize = _prevLayer->layerSize()[0];
	else
		prevLayerSize = _prevLayer->layerSize()[0]*_prevLayer->layerSize()[1]*_prevLayer->layerSize()[2];

	std::vector<float> act = _prevLayer->activations();
	thrust::device_vector<float> input(act.begin(), act.end());
	thrust::device_vector<float> weights(_weights.size());
	if(p == PREDICTION)
		thrust::copy(_weights.begin(), _weights.end(), weights.begin());
	else if(p == TARGET)
		thrust::copy(_cachedWeights.begin(), _cachedWeights.end(), weights.begin());

	thrust::device_vector<float> dotProducts(layerSize);
	thrust::device_vector<float> activations(layerSize);

	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(layerSize), 
		thrust::make_zip_iterator(thrust::make_tuple(dotProducts.begin(), activations.begin())), 
		Dense1dTransform(thrust::raw_pointer_cast(input.data()), thrust::raw_pointer_cast(weights.data()), 
			prevLayerSize, _layerSize[0], _layerSize[1], _layerSize[2]));
	hipDeviceSynchronize();

	// It looks like it works great with std::vector<float> as output vector of thrust::copy. 
	// Maybe try thrust::host_vector<float> as member vectors as well. Also, make sure location of output vector is not changed.
	thrust::copy(dotProducts.begin(), dotProducts.end(), _dotProducts.begin());
	thrust::copy(activations.begin(), activations.end(), _activations.begin());
	
	//std::cout << "DenseLayer::forwardProp" << std::endl;
}

struct Dense1dBack{
	float *_inGrad;
	float *_dotProducts;
	float *_prevAct;
	float *_weights;
	float _delta;
	int _prevLayerSize;

	Dense1dBack(float *ig, float *dp, float *pa, float *w, float d, int pls)
		:_inGrad(ig), _dotProducts(dp), _prevAct(pa), _weights(w), _delta(d), _prevLayerSize(pls)
	{}
	__host__ __device__ thrust::tuple<float, float> operator()(size_t widx)
	{
		// actGrad = gradRelu(inGrad, dotProduct)
		// outGrad = weight*actGrad = weight*gradRelu(inGrad, dotProduct)
		// weightGrad = prevAct*actGrad = prevAct*gradRelu(inGrad, dotProduct) 
		// tdUpdate = weightGrad*delta

		int aidx = widx/_prevLayerSize;

		// actGrad is the gradient of relu activation.
		float actGrad = (_dotProducts[aidx] >= 0)? _inGrad[aidx] : 0;
		float outGrad = _weights[widx]*actGrad;
		float tdUpdate = _prevAct[widx]*actGrad*_delta;

		return thrust::make_tuple(outGrad, tdUpdate);
	}
};


void DenseLayer::backProp(unsigned expNum, const std::vector<double> &action, double delta)
{
	//std::cout << "DenseLayer::backProp" << std::endl;

	std::vector<float> grad((*_vertices).size());
	for(int i=0; i<grad.size(); ++i)
	{
		std::vector<Edge*> outputEdges = (*_vertices)[i]->outputEdges();
		float inGrad = 0;
		for(int j=0; j<outputEdges.size(); ++j)
			inGrad += outputEdges[j]->outGrad();
		
		// In case layer is not output.
		if(outputEdges.size() > 0)
			grad[i] = inGrad;
	}

	thrust::device_vector<float> inGrads(grad.begin(), grad.end());
	thrust::device_vector<float> dotProducts(_dotProducts.begin(), _dotProducts.end());
	//thrust::device_vector<float> actGrad(_activations.size());

	std::vector<float> act = _prevLayer->activations();
	thrust::device_vector<float> prevAct(act.begin(), act.end());
	thrust::device_vector<float> weights(_weights.begin(), _weights.end());

	thrust::device_vector<float> outGrads(_outGrads.size());
	thrust::device_vector<float> tdUpdates(_TDUpdates.size());

	// This works here because outGrads and tdUpdates are of same size.
	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(_weights.size()), 
		thrust::make_zip_iterator(thrust::make_tuple(outGrads.begin(), tdUpdates.begin())), 
		Dense1dBack(thrust::raw_pointer_cast(inGrads.data()), thrust::raw_pointer_cast(dotProducts.data()), 
			thrust::raw_pointer_cast(prevAct.data()), thrust::raw_pointer_cast(weights.data()), delta, prevAct.size()));
	hipDeviceSynchronize();

	thrust::copy(outGrads.begin(), outGrads.end(), _outGrads.begin());
	thrust::copy(tdUpdates.begin(), tdUpdates.end(), _TDUpdates.begin());	

	// prevInGrad = sum over respective outGrad elements. I think this should be done at beginning of backProp for current layer.
	//thrust::device_vector<float> prevInGrad(prevAct.size());
}

void DenseLayer::cacheWeights()
{
	//std::cout << "DenseLayer::cacheWeights" << std::endl;

	_cachedWeights = _weights;
}

unsigned DenseLayer::numHiddenUnits() const
{
	//std::cout << "DenseLayer::numHiddenUnits" << std::endl;	

	return _numHiddenUnits;
}

std::vector<float> DenseLayer::weights() const
{
	std::cout << "DenseLayer::weights" << std::endl;

	return _weights;
}

std::vector<float> DenseLayer::dotProducts() const
{
	std::cout << "DenseLayer::dotProducts" << std::endl;

	return _dotProducts;
}

Tensor1d<Dense1dVertex*>* DenseLayer::vertices() const
{
	//std::cout << "DenseLayer::vertices" << std::endl;
	return _vertices;
}

void DenseLayer::setWeights(const std::vector<float> &w)
{
	std::cout << "DenseLayer::setWeights" << std::endl;

	_weights.clear();
	std::copy(w.begin(), w.end(), std::back_inserter(_weights));
}

